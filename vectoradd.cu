
#include <hip/hip_runtime.h>
__global__ void vectoradd
(
	int size,
	const float* vecA,
	const float* vecB,
	      float* vecC
)
{
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx >= size)
		return;
	
	vecC[idx] = 0;
	for (int i = 0; i < 100000; i++)
		atomicAdd(&vecC[idx], vecA[idx] + vecB[idx]);
}

void vectoradd
(
	hipStream_t stream, 
	int grid_size,
	int block_size,
	int problem_size,
	const float* vecA,
	const float* vecB,
	float* vecC
)
{
	vectoradd<<<grid_size / block_size, block_size, 0, stream>>>(problem_size, vecA, vecB, vecC);
}
